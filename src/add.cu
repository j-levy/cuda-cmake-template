#include "hip/hip_runtime.h"
#include "../include/main_cuda.h"

// function to add the elements of two arrays
__global__ 
void add(int n, float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
    
    // if numblock = (N + blockSize - 1)/blockSize=
    // y[index] = y[index] + x[index];
}
