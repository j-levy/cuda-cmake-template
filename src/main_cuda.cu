#include "hip/hip_runtime.h"
#include "../include/main_cuda.h"
#include "../include/add.h"

int main(void)
{
    int N = 1 << 20; // 1M elements

    float *x = nullptr; // new float[N];
    float *y = nullptr; // new float[N];
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));


    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the GPU
    int blockSize = 128;
    int numBlocks = (N + blockSize - 1) / blockSize;

    // read more on Unified Memory: https://developer.nvidia.com/blog/unified-memory-cuda-beginners/
    // problem with unified memory is, initializing it on CPU moves pages on CPU
    // then accessing on GPU makes a lot of fetch misses.
    // one solution to this is force-prefetching data, like this.


    // Prefetch the data to the GPU
    int device = -1;
    hipGetDevice(&device);
    hipMemPrefetchAsync(x, N*sizeof(float), device, NULL);
    hipMemPrefetchAsync(y, N*sizeof(float), device, NULL);
    add<<<numBlocks, blockSize>>>(N, x, y);

    hipDeviceSynchronize();
    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return maxError;
}